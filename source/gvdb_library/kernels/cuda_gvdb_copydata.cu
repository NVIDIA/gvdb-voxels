#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017, NVIDIA Corporation. 
//
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer 
//    in the documentation and/or  other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may be used to endorse or promote products derived 
//    from this software without specific prior written permission.
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING,
// BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL 
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE 
// OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Version 1.0: Rama Hoetzlein, 5/1/2017
//----------------------------------------------------------------------------------
// File: cuda_gvdb_copydata.cu
//
// GVDB Data Transfers
// - CopyData		3D volume into sub-volume
// - CopyDataZYX	3D volume into sub-volume with ZYX swizzle
// - RetreiveData	3D sub-volume into cuda buffer
// - CopyTexToBuf	2D texture into cuda buffer
// - CopyBufToTex	cuda buffer into 2D texture
//-----------------------------------------------

#include "cuda_math.cuh"

texture<unsigned char, hipTextureType3D, hipReadModeElementType>		volTexInC;
texture<float, hipTextureType3D, hipReadModeElementType>		volTexInF;
surface<void, hipTextureType3D>								volTexOut;

// Zero memory of 3D volume
extern "C" __global__ void kernelFillTex ( int3 res, int dsize, float init_val )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;

	surf3Dwrite ( init_val, volTexOut, t.x*dsize, t.y, t.z );
}

// Copy 3D texture into sub-volume of another 3D texture (char)
extern "C" __global__ void kernelCopyTexC ( int3 offs, int3 res )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;
	uchar val = tex3D ( volTexInC, t.x, t.y, t.z );
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(char), (t.y+offs.y), (t.z+offs.z) );
}

// Copy 3D texture into sub-volume of another 3D texture (float)
extern "C" __global__ void kernelCopyTexF ( int3 offs, int3 res )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;	
	float val = tex3D ( volTexInF, t.x, t.y, t.z );
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(float), (t.y+offs.y), (t.z+offs.z) );
}

// Copy linear memory as 3D volume into sub-volume of a 3D texture
extern "C" __global__ void kernelCopyBufToTexC ( int3 offs, int3 res, uchar* inbuf)
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;	
	unsigned char val = inbuf[ (t.z*res.y + t.y)*res.x + t.x ];	
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(uchar), (t.y+offs.y), (t.z+offs.z) );
}
// Copy linear memory as 3D volume into sub-volume of a 3D texture
extern "C" __global__ void kernelCopyBufToTexF ( int3 offs, int3 res, float* inbuf)
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;	
	float val = inbuf[ (t.z*res.y + t.y)*res.x + t.x ];	
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(float), (t.y+offs.y), (t.z+offs.z) );
}

// Copy 3D texture into sub-volume of another 3D texture with ZYX swizzle (float)
extern "C" __global__ void kernelCopyTexZYX (  int3 offs, int3 res )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;
	float val = tex3D ( volTexInF, t.z, t.y, t.x );
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(float), (t.y+offs.y), (t.z+offs.z) );
}

// Retrieve 3D texture into linear memory (float)
extern "C" __global__ void kernelRetrieveTexXYZ ( int3 offs, int3 src_res, int3 res, float* buf )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= src_res.x || t.y >= src_res.y || t.z >= src_res.z ) return;
	float val = tex3D ( volTexInF, t.x+offs.x, t.y+offs.y, t.z+offs.z );
	buf[ (t.x*res.y + t.y)*res.x + t.z ] = val;
}

// Copy 2D slice of 3D texture into 2D linear buffer
extern "C" __global__ void kernelSliceTexToBufF ( int slice, int3 res, float* outbuf  )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ( x >= res.x || y >= res.y ) return;
	float val = tex3D ( volTexInF, x, y, slice );
	outbuf[ y*res.x + x ] = val;
}

extern "C" __global__ void kernelSliceTexToBufC ( int slice, int3 res, uchar* outbuf  )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ( x >= res.x || y >= res.y ) return;
	uchar val = tex3D ( volTexInC, x, y, slice );
	outbuf[ y*res.x + x ] = val;
}


// Copy 2D linear buffer into the 2D slice of a 3D texture
extern "C" __global__ void kernelSliceBufToTexF ( int slice, int3 res, float* inbuf  )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ( x >= res.x || y >= res.y ) return;
	float val = inbuf[ y*res.x + x ];
	surf3Dwrite ( val, volTexOut, x*sizeof(float), y, slice );
}
extern "C" __global__ void kernelSliceBufToTexC ( int slice, int3 res, uchar* inbuf  )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ( x >= res.x || y >= res.y ) return;
	uchar val = inbuf[ y*res.x + x ];
	surf3Dwrite ( val, volTexOut, x*sizeof(uchar), y, slice );
}


